#include "hip/hip_runtime.h"
ct #include<stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

    __global__ void computeVectors(int *d_arr1, int *d_arr2, double *d_dotProduct, double *d_length1, double *d_length2, int elementsPerThread)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double dot = 0, len1 = 0, len2 = 0;

    for (int i = 0; i < elementsPerThread; i++)
    {
        int index = idx * elementsPerThread + i;
        dot += d_arr1[index] * d_arr2[index];
        len1 += d_arr1[index] * d_arr1[index];
        len2 += d_arr2[index] * d_arr2[index];
    }

    d_dotProduct[idx] = dot;
    d_length1[idx] = len1;
    d_length2[idx] = len2;
}

void generateRandomArray(int *array, int size)
{
    for (int i = 0; i < size; i++)
    {
        array[i] = rand() % 10000;
    }
}

int main(int argc, char **argv)
{
    if (argc < 3)
    {
        printf("Usage: %s <array_size> <threads_per_block>\n", argv[0]);
        return -1;
    }

    int arraySize = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int blocks = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    int elementsPerThread = arraySize / (threadsPerBlock * blocks);

    // Host memory allocation
    int *h_arr1 = (int *)malloc(arraySize * sizeof(int));
    int *h_arr2 = (int *)malloc(arraySize * sizeof(int));
    double *h_dotProduct, *h_length1, *h_length2;
    h_dotProduct = (double *)malloc(blocks * threadsPerBlock * sizeof(double));
    h_length1 = (double *)malloc(blocks * threadsPerBlock * sizeof(double));
    h_length2 = (double *)malloc(blocks * threadsPerBlock * sizeof(double));

    // Initialize arrays
    srand(time(NULL));
    generateRandomArray(h_arr1, arraySize);
    generateRandomArray(h_arr2, arraySize);

    // Device memory allocation
    int *d_arr1, *d_arr2;
    double *d_dotProduct, *d_length1, *d_length2;
    hipMalloc((void **)&d_arr1, arraySize * sizeof(int));
    hipMalloc((void **)&d_arr2, arraySize * sizeof(int));
    hipMalloc((void **)&d_dotProduct, blocks * threadsPerBlock * sizeof(double));
    hipMalloc((void **)&d_length1, blocks * threadsPerBlock * sizeof(double));
    hipMalloc((void **)&d_length2, blocks * threadsPerBlock * sizeof(double));

    // Transfer data to device
    hipMemcpy(d_arr1, h_arr1, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, h_arr2, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Kernel launch
    computeVectors<<<blocks, threadsPerBlock>>>(d_arr1, d_arr2, d_dotProduct, d_length1, d_length2, elementsPerThread);
    hipDeviceSynchronize();

    // Transfer results back to host
    hipMemcpy(h_dotProduct, d_dotProduct, blocks * threadsPerBlock * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_length1, d_length1, blocks * threadsPerBlock * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_length2, d_length2, blocks * threadsPerBlock * sizeof(double), hipMemcpyDeviceToHost);

    // Combine results on host
    double dotProduct = 0, length1 = 0, length2 = 0;
    for (int i = 0; i < blocks * threadsPerBlock; i++)
    {
        dotProduct += h_dotProduct[i];
        length1 += h_length1[i];
        length2 += h_length2[i];
    }

    length1 = sqrt(length1);
    length2 = sqrt(length2);
    double cosine = dotProduct / (length1 * length2);
    double angle = acos(cosine) * 180.0 / M_PI;

    // Output results
    printf("Dot product: %.2f\n", dotProduct);
    printf("Cosine of angle: %.4f\n", cosine);
    printf("Angle (degrees): %.2f\n", angle);

    // Free memory
    free(h_arr1);
    free(h_arr2);
    free(h_dotProduct);
    free(h_length1);
    free(h_length2);
    hipFree(d_arr1);
    hipFree(d_arr2);
    hipFree(d_dotProduct);
    hipFree(d_length1);
    hipFree(d_length2);

    return 0;
}
